
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int a, int b, int *c) {
    *c = a + b;
}

int main(void) {
    int c;
    int *dev_c;

    hipMalloc( (void **) &dev_c, sizeof(int)); // fails here, returning 46

    add<<<1,1>>>(2,7,dev_c);
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    printf("hello world, %d\n",c);
    hipFree( dev_c);
    return 0;
}
